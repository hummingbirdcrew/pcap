#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Corrected the function signature to accept the array size 'n'
__global__ void vectoraddthreadn(int* a, int* b, int* r, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    // Use correct variable 'i' to access elements
    if (i < n) {
        r[i] = a[i] + b[i];
    }
}

// Corrected the function signature to accept the array size 'n'
__global__ void vectoraddblockn(int* a, int* b, int* r, int n) {
    int i = threadIdx.x;
    // Use correct variable 'i' to access elements
    if (i < n) {
        r[i] = a[i] + b[i];
    }
}

int main() {
    int* a, * b, * c; // Changed 'r' to 'c'
    int n=7;
    a = (int*)malloc(n * sizeof(int));
    b = (int*)malloc(n * sizeof(int));
    c = (int*)malloc(n * sizeof(int)); // Changed 'r' to 'c'
    for (int i = 0; i <n; ++i) {
        a[i] = i;
        b[i] = 2 * i;
    }
    printf("pirnting a");
    for (int i = 0; i <n; ++i) {
        printf("%d",a[i]);
        printf("\n");

    }
    printf("pirnting b");
    for (int i = 0; i <n; ++i) {
        printf("%d",b[i]);
        printf("\n");

    }

    int* d_a, * d_b, * d_c; // Changed 'd_r' to 'd_c'
    hipMalloc((void**)&d_a, n * sizeof(int));
    hipMalloc((void**)&d_b, n * sizeof(int));
    hipMalloc((void**)&d_c, n * sizeof(int)); // Changed 'd_r' to 'd_c'

    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    // Pass the size 'n' to the kernel function
    vectoraddblockn<<<1, n>>>(d_a, d_b, d_c, n);

    // Copy result back to host
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    printf("Result using block size as N:\n");
    for (int i = 0; i < n; ++i) {
        printf("%d ", c[i]);
    }
    printf("\n\n");

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Pass the size 'n' to the kernel function
    vectoraddthreadn<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    // Copy result back to host
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    printf("Result using N threads:\n");
    for (int i = 0; i < n; ++i) {
        printf("%d ", c[i]);
    }
    printf("\n");

    // Free allocated memory on the device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free allocated memory on the host
    free(a);
    free(b);
    free(c);

    return 0;
}
