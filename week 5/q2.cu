#include "hip/hip_runtime.h"
###Implement a CUDA program to add two vectors of length N by keeping the number of threads per block as 256 (constant) and vary the number of blocks to handle N elements


%%cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void vectoradd(int* a,int* b,int* c,int n){
    int tid=threadIdx.x+blockIdx.x*blockDim.x;
    if(tid<n){
    c[tid]=a[tid]+b[tid];
    }
}

int main(){
    int *a,*b,*c;
    int *da,*db,*dc;
    int n=7;
    a=(int*)malloc(n*sizeof(int));
    b=(int*)malloc(n*sizeof(int));
    c=(int*)malloc(n*sizeof(int));

    for (int i = 0; i <n; ++i) {
        a[i] = i;
        b[i] = 2 * i;
    }
    printf("pirnting a\n");
    for (int i = 0; i <n; ++i) {
        printf("%d",a[i]);
        printf("\n");

    }
    printf("pirnting b\n");
    for (int i = 0; i <n; ++i) {
        printf("%d",b[i]);
        printf("\n");

    }

    hipMalloc((void**)&da,n*sizeof(int));
    hipMalloc((void**)&db,n*sizeof(int));
    hipMalloc((void**)&dc,n*sizeof(int));

    hipMemcpy(da,a,n*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(db,b,n*sizeof(int),hipMemcpyHostToDevice);

    // Calculate the number of blocks needed
    int numBlocks = (n +255) /256;

    vectoradd<<<numBlocks,256>>>(da,db,dc,n);

    hipMemcpy(c,dc,n*sizeof(int),hipMemcpyDeviceToHost);

    printf("Result using N threads:\n");
    for (int i = 0; i < n; ++i) {
        printf("%d ", c[i]);
    }
    printf("\n");

    // Free allocated memory on the device
    hipFree(da);
    hipFree(db);
    hipFree(dc);

    // Free allocated memory on the host
    free(a);
    free(b);
    free(c);

    return 0;
}
