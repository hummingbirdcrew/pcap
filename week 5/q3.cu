#include "hip/hip_runtime.h"
#####third question
Write a program in CUDA which performs convolution operation on one dimensional input array N of size width using a mask array M of size mask_width to produce the resultant one dimensional array P of size width


%%cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void convolution(int* p, int* m, int* r, int ml, int pl) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int rvalue = 0;
    int pstart = tid - (ml) / 2;
    for (int j = 0; j < ml; j++) {
        if ((pstart + j) >= 0 && (pstart + j) < pl) {
            rvalue += p[pstart + j] * m[j];
        }
    }
    r[tid] = rvalue;
}

int main() {
    int *m, *n, *r;
    int *dm, *dn, *dr;
    int ns = 7;
    int ms = 5;

    m = (int*)malloc(ms * sizeof(int));
    n = (int*)malloc(ns * sizeof(int));
    r = (int*)malloc(ns * sizeof(int));

    for (int i = 0; i < ns; ++i) {
        n[i] = i;
    }
    for (int i = 0; i < ms; ++i) {
        m[i] = i * 2;
    }
    printf("printing array n\n");
    for (int i = 0; i < ns; ++i) {
        printf("%d ", n[i]);
    }
    printf("\n");

    printf("printing mask m\n");
    for (int i = 0; i < ms; ++i) {
        printf("%d ", m[i]);
    }
    printf("\n");

    hipMalloc((void**)&dn, ns * sizeof(int));
    hipMalloc((void**)&dm, ms * sizeof(int));
    hipMalloc((void**)&dr, ns * sizeof(int));

    hipMemcpy(dn, n, ns * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dm, m, ms * sizeof(int), hipMemcpyHostToDevice);

    convolution<<<(ns + 4 - 1) / 4, 4>>>(dn, dm, dr, ms, ns);

    hipMemcpy(r, dr, ns * sizeof(int), hipMemcpyDeviceToHost);

    printf("Result after convolution:\n");
    for (int i = 0; i < ns; ++i) {
        printf("%d ", r[i]);
    }
    printf("\n");

    hipFree(dn);
    hipFree(dm);
    hipFree(dr);
    free(m);
    free(n);
    free(r);

    return 0;
}
