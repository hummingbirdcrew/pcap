#include "hip/hip_runtime.h"
####fourth question Write a program in CUDA to process a 1D array containing angles in radians to generate sine of the angles in the output array. Use appropriate function.


%%cu
#include <stdio.h>
#include <math.h>

#define N 5 // Length of the array
#define THREADS_PER_BLOCK 256

__global__ void computeSine(float *input, float *output, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        output[tid] = sinf(input[tid]);
    }
}

int main() {
    float *h_input, *h_output; // Host arrays
    float *d_input, *d_output; // Device arrays

    // Allocate memory on the host
    h_input = (float*)malloc(N * sizeof(float));
    h_output = (float*)malloc(N * sizeof(float));

    // Initialize host input array with angles in radians
    for (int i = 0; i < N; i++) {
        h_input[i] = static_cast<float>(i) * (3.14159f / 180.0f); // Convert degrees to radians
    }

    // Allocate memory on the device
    hipMalloc((void**)&d_input, N * sizeof(float));
    hipMalloc((void**)&d_output, N * sizeof(float));

    // Copy host input array to device
    hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice);

    // Calculate the number of blocks needed
    int numBlocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // Launch the kernel
    computeSine<<<numBlocks, THREADS_PER_BLOCK>>>(d_input, d_output, N);

    // Copy result back to host
    hipMemcpy(h_output, d_output, N * sizeof(float), hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < N; i++) {
        printf("sin(%.2f radians) = %.6f\n", h_input[i], h_output[i]);
    }

    // Free memory on the device
    hipFree(d_input);
    hipFree(d_output);

    // Free memory on the host
    free(h_input);
    free(h_output);

    return 0;
}